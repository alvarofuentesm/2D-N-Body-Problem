#include <iostream>
#include <hip/hip_runtime.h>
#include<cmath>


const double NEWTON_G = 6.67384e-11;
const double SOFTENING =  1e-9f;

__constant__ double NEWTON_GG = 6.67384e-11;
__constant__ double SOFTENINGG =  1e-9f;

void writeSoA(double** f, int B, int size, const char *filename){
	FILE* file;
    file=fopen(filename, "w");
	
	fprintf(file, "%d\n", B);
	for (int i = 0; i < 5; i++){
		for (int j = 0; j < size/5; j++){
			fprintf(file, "%lf ", ((*f)[j + i*(size/5)]));
		}
		fprintf(file, "\n");
	}
	fclose(file);
}

void ReadSoA(double** f, int* B, const char *filename) {
    FILE *fp;
    fp = fopen(filename, "r");
    fscanf(fp, "%d", B);
    int size = (*B) * 5; // 5 atributos: masa, pos_x, pos_y, vel_x, vel_y
    
    double* F = new double[size];
    int i;
    for (i = 0; i < size; i++) {
    	fscanf(fp, "%lf ", &(F[i]));
    }
    *f = F;
    fclose(fp);
}


void printArray(int size, double *arr) {
    printf("[");
	for (int i = 0; i < size; i++) {
		printf("%lf ", arr[i]);
	}
    printf("]\n");
}


void N_body_CPU(int size, double delta_t, double *f, double *fout, int T){
    //printf("N_body_CPU\n");
    
    for (int body_i = 0; body_i < size/5; body_i++){ // para cada cuerpo
        //printf("body %d ", body_i);
        //if (body_i == 95 && T == 0) printf("(%lf, %lf, %lf, %lf, %lf)\n", f[body_i], f[body_i + (size/5)], f[body_i + (size/5)*2], f[body_i + (size/5)*3], f[body_i + (size/5)*4]);
        double mass1  = f[body_i];
        double x1 = f[body_i + (size/5)];
        double y1 = f[body_i + (size/5)*2];
        double vx1 = f[body_i + (size/5)*3];
        double vy1 = f[body_i + (size/5)*4];

    
        double Fx = 0;
        double Fy = 0;
        for (int j = 0; j < size/5; j++){ // comparar con otros cuerpos
            if (j == body_i) continue; // creo que puedo obviarlo pues el radio seria cero (aunque nos da division por cero)
            double mass2  = f[j];
            double x2 = f[j + (size/5)];
            double y2 = f[j + (size/5)*2];


            double distance =  sqrt( pow(x2-x1, 2) + pow(y2-y1, 2) + pow(SOFTENING, 2) ); 
            //if (body_i == 0) printf("distance: %lf\n", distance);
            double angle = atan((y2-y1)/(x2-x1));

            Fx +=  NEWTON_G*mass1*mass2/(pow(distance, 2)) * cos(angle); 
            Fy +=  NEWTON_G*mass1*mass2/(pow(distance, 2)) * sin(angle); 
            
        }
        

        double new_vx1 = vx1 + Fx*delta_t/mass1;
        double new_vy1 = vy1 + Fy*delta_t/mass1;

    
        // a futuro, usar otro arreglo para la masa, pues no cambia
        fout[body_i] = mass1; 
        fout[body_i + (size/5)]   = x1 + new_vx1*delta_t; //new x
        fout[body_i + (size/5)*2] = y1 + new_vy1*delta_t ; //new  y
        fout[body_i + (size/5)*3] = new_vx1; //new vx
        fout[body_i + (size/5)*4] = new_vy1; //new vy
    }
}

__global__ void N_body_GPU(int size, double delta_t, double *f, double *fout){
    int body_i= threadIdx.x + blockDim.x*blockIdx.x;
    if (body_i < size/5){

        double mass1  = f[body_i];
        double x1 = f[body_i + (size/5)];
        double y1 = f[body_i + (size/5)*2];
        double vx1 = f[body_i + (size/5)*3];
        double vy1 = f[body_i + (size/5)*4];
        double mass2,x2,y2,distance,angle,new_vx1,new_vy1;
        double Fx = 0;
        double Fy = 0;
        for (int j = 0; j < size/5; j++){ // comparar con otros cuerpos
                if (j != body_i){
                mass2  = f[j];
                x2 = f[j + (size/5)];
                y2 = f[j + (size/5)*2];

                distance =  sqrt( (x2-x1)*(x2-x1) + (y2-y1)*(y2-y1) + SOFTENINGG); 
                //printf("distance: %lf\n", distance);
                angle = atan((y2-y1)/(x2-x1));

                Fx +=  NEWTON_GG*mass2/(pow(distance, 2)) * cos(angle); 
                Fy +=  NEWTON_GG*mass2/(pow(distance, 2)) * sin(angle); 
            }
        }
        //printf("F: %lf\n", F);

        new_vx1 = vx1 + Fx*delta_t;
        new_vy1 = vy1 + Fy*delta_t;

        //printf("F*delta_t/mass1: %lf\n", F*delta_t/mass1);
        //printf("new_vx1: %lf\n", new_vx1);
        //printf("new_vy1: %lf\n", new_vy1);
    
        // a futuro, usar otro arreglo para la masa, pues no cambia
        fout[body_i] = mass1; 
        fout[body_i + (size/5)]   = x1 + new_vx1*delta_t; //new x
        fout[body_i + (size/5)*2] = y1 + new_vy1*delta_t ; //new  y
        fout[body_i + (size/5)*3] = new_vx1; //new vx
        fout[body_i + (size/5)*4] = new_vy1; //new vy
    }
}


__global__ void N_body_GPU_F(int size, double delta_t, double *f, double *fout,int T){
	int body_i= threadIdx.x + blockDim.x*blockIdx.x;
	if (body_i<size/5){
		extern __shared__ double datos[];
		// 5 atributos: masa
		datos[body_i			]= f[body_i];
		datos[body_i+ (size/5)	]= f[body_i + (size/5)];
       	datos[body_i+ (size/5)*2]= f[body_i + (size/5)*2];
		double autx,auty,rx,ry,vx,vy;
        vx=f[body_i+ (size/5)*3];
        vy=f[body_i+ (size/5)*4];
        double angle;
        double Ax,Ay;
		fout[body_i]=datos[body_i];
        for (int t = 0; t < T; ++t){
            __syncthreads();
            Ax=0.0,Ay=0.0;
            autx=datos[body_i+ (size/5)  ];
            auty=datos[body_i+ (size/5)*2];
            auty=datos[body_i+ (size/5)*2];
    		for (int i = 0; i < size/5; ++i){
                if (i!=body_i){
        			rx=autx-datos[i+ (size/5)  ];
        			ry=auty-datos[i+ (size/5)*2];
        			angle=atan(ry/rx);
                    rx=datos[i]/sqrt(rx*rx+ry*ry+SOFTENINGG);
        			Ax += rx*cos(angle);
        			Ay += rx*sin(angle);
                }
    		}
    		Ax*=NEWTON_GG*delta_t;
    		Ay*=NEWTON_GG*delta_t;
    		datos[body_i+ (size/5)  ]=autx+vx*delta_t+Ax*delta_t;
    		datos[body_i+ (size/5)*2]=auty+vy*delta_t+Ay*delta_t;
    		vx=Ax+vx;
    		vy=Ay+vy;
        }
        fout[body_i+ (size/5)  ]=autx+vx*delta_t+Ax*delta_t;
        fout[body_i+ (size/5)*2]=auty+vy*delta_t+Ay*delta_t;
        fout[body_i+ (size/5)*3]=Ax+vx;
        fout[body_i+ (size/5)*4]=Ay+vy;
	}
}

int main() {	


    hipEvent_t ct1, ct2, ct3, ct4;

    clock_t t1, t2;
    double ms;

    char filename[] = "input.txt";
    char filename_out[] = "-CPU-Resultado.txt";
    char filename_aux[30];

    char final[] = "final";
    char directory[] = "data/";
    char directory_aux[30];

	float dt,dt2;

	//int iterator=2;
	int B;
	
    double *f, *fout, *fhost, *fhostout, *faux,*ff;

    int grid_size, block_size = 256;

	ReadSoA(&fhost, &B, filename);
    int size = B*5;

    hipMalloc((void**)&f, size* sizeof(double));
    hipMalloc((void**)&ff, size* sizeof(double));
    hipMalloc((void**)&fout, size* sizeof(double));

    hipMemcpy(f, fhost, size* sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(ff, fhost, size* sizeof(double), hipMemcpyHostToDevice);
    int debug = 0;
    
    if (debug){
        printf("B: %d\n",  B );
        printf("size: %d\n", size);
        printArray(size, fhost);
    } 

    double delta_t = 0.01;

    fhostout = new double[size];

    if (debug){
        printArray(size, fhost);
    }

    long T = 10000;
    
    char integer_string[32];
    char integer_string2[32];

    /*** CPU ***/
    int cpu = 1;
    if (cpu){ 

        t1 = clock();
        for (long t = 0; t < T; t++){
            N_body_CPU(size, delta_t, fhost, fhostout, t);
            faux=fhost;
            fhost = fhostout;
            fhostout=faux;
            
            if (t % 1000 == 0 || t == T-1){
                sprintf(integer_string, "%d", t);
                sprintf(integer_string2, "-%d", T);
                strcpy(filename_aux, filename_out);
                strcpy(directory_aux, directory);

                writeSoA(&fhostout, B, size, 
                          strcat(directory_aux, strcat(integer_string, strcat(integer_string2, filename_aux) ) ));
            }
            
            //printArray(size, fhostout);
            //std::cout << "-----------------------" << std::endl;
        }
        t2 = clock();

        if (debug){
            printArray(size, fhost);
        }

        ms = 1000.0 * (double)(t2 - t1) / CLOCKS_PER_SEC;
    	std::cout << "Tiempo CPU  : " << ms << "[ms]" << std::endl;

        //writeSoA(&fhostout, B, size, strcat(final, filename_out) );
    }
    int long_simulation = 0;
    
    if (long_simulation){
        T = 20000*20;
    }


    /***  GPU ***/
    int gpu1 = 1;
    if (gpu1){

        char filename_outGPU[] = "-GPU-Resultado.txt";
        grid_size = (int)ceil((float) B / block_size);

        hipEventCreate(&ct1);
        hipEventCreate(&ct2);
        hipEventRecord(ct1);
        for (long t = 0; t < T; t++){
            N_body_GPU<<<grid_size, block_size>>>(size, delta_t, f, fout);
            faux = fout;
            fout = f;
            f = faux;
            
            if (t % 1000 == 0 || t == T-1){
                sprintf(integer_string, "%d", t);
                sprintf(integer_string2, "-%d", T);
                strcpy(filename_aux, filename_outGPU);
                strcpy(directory_aux, directory);
                hipMemcpy(fhostout, f, size* sizeof(double), hipMemcpyDeviceToHost);

                writeSoA(&fhostout, B, size, 
                            strcat(directory_aux, strcat(integer_string, strcat(integer_string2, filename_aux))) );
            }
        }
        hipEventRecord(ct2);
        hipEventSynchronize(ct2);
        hipEventElapsedTime(&dt, ct1, ct2);

        std::cout << "Tiempo GPU  : " << dt << "[ms]" << std::endl;

        hipMemcpy(fhostout, f, size* sizeof(double), hipMemcpyDeviceToHost);
        //strcpy(filename_out, "GPU-Resultado.txt");
        //writeSoA(&fhostout, B, size, filename_outGPU);
    }

    /***  GPU Fast ***/
    char filename_outFPU[] = "data/FPU-Resultado.txt";

    grid_size = (int)ceil((float) B / block_size);

    hipEventCreate(&ct3);
    hipEventCreate(&ct4);
    hipEventRecord(ct3);

    N_body_GPU_F<<<grid_size, block_size,B*3* sizeof(double)>>>(size, delta_t, ff, fout,T);

    hipEventRecord(ct4);
    hipEventSynchronize(ct4);
    hipEventElapsedTime(&dt2, ct3, ct4);

    std::cout << "Tiempo GPU-F: " << dt2 << "[ms]" << std::endl;

    hipMemcpy(fhostout, fout, size* sizeof(double), hipMemcpyDeviceToHost);

    writeSoA(&fhostout, B, size, filename_outFPU);
    

    delete[] fhostout;

}
